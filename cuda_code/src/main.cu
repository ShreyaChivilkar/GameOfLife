#include "hip/hip_runtime.h"
/*
Author: Shreya Chivilkar
Last Date Modified: 8th November 2024
Description: CUDA-based John Conway’s Game of Life -> This file "main.cpp" contains code that is used to generate the the grid, check if the cell will be alive or dead based on neighbors, replace the grid with the new grid.
I have developed functions for the three types of memory mentioned - Normal, Pinned, Managed 
*/

#include <SFML/Graphics.hpp>
#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

//Defined the default values of parameters
const int DEFAULT_THREADS = 32;
const int DEFAULT_CELL_SIZE = 5;
const int DEFAULT_WIDTH = 800;
const int DEFAULT_HEIGHT = 600;
const int UPDATES_PER_BATCH = 100;

//This is the code for the kernel function - it is used for counting live neighbors and updating cells based on the neighbours
__global__ void updateGridKernel(int* logicGrid, int* newGrid, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < rows * cols) {
        int liveNeighbors = 0;
        int row = x / cols;
        int col = x % cols;

        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                //This will skip the current cell
                if (i == 0 && j == 0) continue;
                int nx = (row + i + rows) % rows;
                int ny = (col + j + cols) % cols;
                liveNeighbors += logicGrid[nx * cols + ny];
            }
        }

        /*Applying the game of life rules:
        Checked the count of live neighbours for the current cell
        If the cell is alive - it will become dead when live neighbors are less than 2 or greater than 3, otherwise it will remain alive
        Similarly, if the cell is dead - it will become alive only if the live neighbours count is 3, otherwise it will remain dead
        */
        int currentState = logicGrid[x];
        if (currentState == 1 && (liveNeighbors < 2 || liveNeighbors > 3)) {
            newGrid[x] = 0;  
        } else if (currentState == 0 && liveNeighbors == 3) {
            newGrid[x] = 1; 
        } else {
            newGrid[x] = currentState;
        }
    }
}

void updateGridGPUNormal(int* logicGrid, int rows, int cols, int numThreads) {
    int gridSize = rows * cols * sizeof(int);
    int *d_logicGrid, *d_newGrid;

    //Allocate the memory 
    hipMalloc(&d_logicGrid, gridSize);
    hipMalloc(&d_newGrid, gridSize);

    //Copy the grid values from the CPU to GPU (Host to Device)
    hipMemcpy(d_logicGrid, logicGrid, gridSize, hipMemcpyHostToDevice);

    //Assign the dimensions
    dim3 blockDim(numThreads);
    dim3 gridDim((rows*cols + blockDim.x - 1) / blockDim.x);

    //Update the grid
    for (int i = 0; i < UPDATES_PER_BATCH; ++i) {
        updateGridKernel<<<gridDim, blockDim>>>(d_logicGrid, d_newGrid, rows, cols);
        hipDeviceSynchronize();
        std::swap(d_logicGrid, d_newGrid);
    }
    
    // Copy the Updated grid from GPU to CPU (Device to Host)
    hipMemcpy(logicGrid, d_logicGrid, gridSize, hipMemcpyDeviceToHost);
    
    // Free the allocated memory
    hipFree(d_logicGrid);
    hipFree(d_newGrid); 
}

void updateGridGPUPinned(int* logicGrid, int rows, int cols, int numThreads) {
    int gridSize = rows * cols * sizeof(int);
    int *d_logicGrid, *d_newGrid, *h_logicGrid;

    //Allocate the memory 
    hipHostAlloc(&h_logicGrid, gridSize, hipHostMallocDefault);
    hipMalloc(&d_logicGrid, gridSize);
    hipMalloc(&d_newGrid, gridSize);

    //Copy the grid values from cpu to pinned memory of cpu (Host to Host)
    hipMemcpy(h_logicGrid, logicGrid, gridSize, hipMemcpyHostToHost);
    //Copy the grid values from pinned memory of cpu to GPU (Host to Device)
    hipMemcpy(d_logicGrid, h_logicGrid, gridSize, hipMemcpyHostToDevice);

    //Assign the dimensions
    dim3 blockDim(numThreads);
    dim3 gridDim((rows * cols + blockDim.x - 1) / blockDim.x);

    //Update the grid
    for (int i = 0; i < UPDATES_PER_BATCH; ++i) {
        updateGridKernel<<<gridDim, blockDim>>>(d_logicGrid, d_newGrid, rows, cols);
        hipDeviceSynchronize();
        std::swap(d_logicGrid, d_newGrid);
    }
    
    // Copy the updated grid values from GPU to CPU (Device to host)
    hipMemcpy(logicGrid, d_logicGrid, gridSize, hipMemcpyDeviceToHost);
    
    // Free the allocated memory
    hipHostFree(logicGrid);
    hipFree(d_newGrid);
    hipFree(d_logicGrid);
}

void updateGridGPUManaged(int* logicGrid, int rows, int cols, int numThreads) {
    int gridSize = rows * cols * sizeof(int);
    int *d_logicGrid, *d_newGrid;

    //Allocate the memory 
    hipError_t error = hipMallocManaged(&d_logicGrid, gridSize);
    if(error!=hipSuccess){
        std::cerr<<"Error for memory allocation: "<<hipGetErrorString(error)<<std::endl;
        return;
    }
    hipMemcpy(d_logicGrid, logicGrid, gridSize, hipMemcpyHostToDevice);

    error = hipMallocManaged(&d_newGrid, gridSize);
    if(error!=hipSuccess){
        std::cerr<<"Error for memory allocation: "<<hipGetErrorString(error)<<std::endl;
        return;
    }
    
    // Set the dimensions
    dim3 blockDim(numThreads);
    dim3 gridDim((rows * cols + blockDim.x - 1) / blockDim.x);
    
    //Update the grid
    for (int i = 0; i < UPDATES_PER_BATCH; ++i) {
        updateGridKernel<<<gridDim, blockDim>>>(d_logicGrid, d_newGrid, rows, cols);
        hipDeviceSynchronize();
        std::swap(d_logicGrid, d_newGrid);
    }
    
    // Copy the updated grid values from GPU to CPU (Device to host)
    error = hipMemcpy(logicGrid, d_logicGrid, gridSize, hipMemcpyDeviceToHost);

    if(error!=hipSuccess){
        std::cerr<<"Error during memory copy: "<<hipGetErrorString(error)<<std::endl;
        return;
    }
    
    // Free the allocated memory
    hipFree(d_logicGrid);
    hipFree(d_newGrid);
    
}

int main(int argc, char* argv[]) {
    //Set default values for the parameters
    int numThreads = DEFAULT_THREADS;
    int cellSize = DEFAULT_CELL_SIZE;
    int window_width = DEFAULT_WIDTH;
    int window_height = DEFAULT_HEIGHT;
    std::string processingType = "normal";

     // Handle the input params and validate based on conditions
    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];

        if (arg == "-n" && i + 1 < argc) {
            numThreads = std::stoi(argv[++i]);
            if (numThreads < 32) {
                std::cerr << "Number of threads should be greater than 32, setting to default value\n";
                return 1;
            }
        } else if (arg == "-c" && i + 1 < argc) {
            cellSize = std::stoi(argv[++i]);
            if (cellSize < 1) {
                std::cerr << "Cell size must be greater than or equal to 1.\n";
                return 1;
            }
        } else if (arg == "-x" && i + 1 < argc) {
            window_width = std::stoi(argv[++i]);
        } else if (arg == "-y" && i + 1 < argc) {
            window_height = std::stoi(argv[++i]);
        } else if (arg == "-t" && i + 1 < argc) {
            processingType = argv[++i];
        }
    }

    int gridWidth = window_width / cellSize;
    int gridHeight = window_height / cellSize;

    // Initialise the logic grid
    std::vector<int> logicGrid(gridHeight * gridWidth, 0);
    for (int i = 0; i < gridHeight * gridWidth; ++i) {
        logicGrid[i] = (std::rand() % 2);
    }

    sf::RenderWindow window(sf::VideoMode(window_width, window_height), "CUDA-based John Conway’s Game of Life");

    
    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed || sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) window.close();
        }
        auto startTime = std::chrono::high_resolution_clock::now();
        //processing based on type selected

       
        if(processingType=="managed"){
            updateGridGPUManaged(logicGrid.data(), gridHeight, gridWidth, numThreads);
        }
        else if(processingType=="pinned"){
            updateGridGPUPinned(logicGrid.data(), gridHeight, gridWidth, numThreads);
        }
        else{
            updateGridGPUNormal(logicGrid.data(), gridHeight, gridWidth, numThreads);
        }
        //Calculate the time for 100 iterations
        auto endTime = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);

        std::cout << UPDATES_PER_BATCH << " generations took " << duration.count() << " microsecs with " << numThreads << " threads per block using "
                  << processingType << " memory allocation"<<std::endl;
        window.clear();


        for (int i = 0; i < gridHeight; ++i) {
            for (int j = 0; j < gridWidth; ++j) {
                sf::RectangleShape cell(sf::Vector2f(cellSize, cellSize));
                cell.setPosition(j * cellSize, i * cellSize);
                cell.setFillColor(logicGrid[i * gridWidth + j] ? sf::Color::White : sf::Color::Black);
                window.draw(cell);
            }
        }

        window.display();
    }
    return 0;
}
